
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

__global__ void VecAdd(float* A, float* B, float*
    C, int N_op,int op_loop){

    // N_op : no of total ops
    // op_loop: no of ops to do in a loop
    // Host code
    int j;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N_op){
        for (j=0;j<op_loop;j++){
            C[i*op_loop+j] = A[i*op_loop+j] + B[i*op_loop+j];
        }
    }

    }

int main() {
    int N = pow(2,15);
    
    int threadsPerBlock_op=256;
    int avg_runs=1000;

    size_t size = N * sizeof(float);

    int loop;
    int op_loop;
    int op_loop_array[10];
    int op_loop_ii;
    int clock_loop;
    float time_spent;

    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize input vectors
    printf("Array A (first 10 values) \n ");
    for(loop = 0; loop < N; loop++){
    h_A[loop] = rand() % 100 + 1;
    if (loop<10){
        printf("%f ", h_A[loop]);
    }
    }

    printf("\nArray B (first 10 values) \n ");
    for(loop = 0; loop < N; loop++){
        h_B[loop] = rand() % 100 + 1;
        if (loop<10){
            printf("%f ", h_B[loop]);
    }
    }

    for (op_loop_ii=0;op_loop_ii<10;op_loop_ii++){
        op_loop_array[op_loop_ii]=pow(2,op_loop_ii);
    }

    // Allocate vectors in device memory
    float* d_A;   hipMalloc(&d_A, size);
    float* d_B;   hipMalloc(&d_B, size);
    float* d_C;   hipMalloc(&d_C, size);

    //GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size,hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size,hipMemcpyHostToDevice);

    for (op_loop_ii=0;op_loop_ii<10;op_loop_ii++){
        op_loop=op_loop_array[op_loop_ii];

        for(clock_loop=0;clock_loop<avg_runs;clock_loop++){

        if (clock_loop==1){
            hipEventRecord(start, 0);
        }

        //ops per loop
        //printf("Ops per loop %d",op_loop);

        // Invoke kernel
        int threadsPerBlock = threadsPerBlock_op;
        int N_op=(N + op_loop -1)/op_loop;
        int blocksPerGrid = (N_op + threadsPerBlock - 1) /threadsPerBlock;
        VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A,d_B, d_C, N_op,op_loop);

        // h_C contains the result in host memory
        hipMemcpy(h_C, d_C, size,hipMemcpyDeviceToHost);

        //printf("\nArray C (first 10 outputs)\n");
        //for(loop = 0; loop < 10; loop++)
        //printf("%f ", h_C[loop]);
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_spent, start, stop);
        time_spent=time_spent/(avg_runs-1)*10;

        printf("\n Average Time spent in loop %d is %f",op_loop,time_spent);
    }
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    // Free host memory
    free(h_A);
    free(h_B);
    return 0;
}