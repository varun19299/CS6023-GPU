#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

void fill_matrix(double *mat, unsigned numRows, unsigned numCols)
{
    for(unsigned i=0; i < numRows; i++)
       for(unsigned j=0; j < numCols; j++)
       {
         mat[i*numCols + j] = i*2.1f + j*3.2f;
       }
}

void print_matrix_to_file(double *mat, unsigned numRows, unsigned numCols)
{
  const char *fname = "assignment2_1_out";
  FILE *f = fopen(fname, "w");
  for(unsigned i=0; i < numRows; i++)
  {
     for(unsigned j=0; j < numCols; j++)
     fprintf(f,"%4.4f ", mat[i*numCols + j]);
     fprintf(f,"\n");
}
fclose(f); }


__global__ void MatrixMulKernel_row_maj(float* M, float* N, float* P, int Width) { 
    // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x+threadIdx.x; 
    
    if ((Row < Width) && (Col < Width)) {
            float Pvalue = 0;
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[Row*Width+k]*N[k*Width+Col];
        }
            P[Row*Width+Col] = Pvalue;
        }
    }

    __global__ void MatrixMulKernel_col_maj(float* M, float* N, float* P, int Width) { 
        // Calculate the row index of the P element and M
        int Row = blockIdx.y*blockDim.y+threadIdx.x;
        // Calculate the column index of P and N
        int Col = blockIdx.x*blockDim.x+threadIdx.y; 
        
        if ((Row < Width) && (Col < Width)) {
                float Pvalue = 0;
            for (int k = 0; k < Width; ++k) {
                Pvalue += M[Row*Width+k]*N[k*Width+Col];
            }
                P[Row*Width+Col] = Pvalue;
            }
        }

int main(int argc,char **argv) {
    int N = 8192;
    size_t size = N *N* sizeof(double);

    double*h_matA = (double*)malloc(size);
    double*h_matB = (double*)malloc(size);
    double*h_matC = (double*)malloc(size); // result

    int loop1; int loop2; // loop variables

    fill_matrix(h_matA,N,N);
    fill_matrix(h_matB,N,N);

    printf("\nMatrix A (first 10*10 inputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", h_matA[loop1][loop2]);
    }

    printf("\nMatrix B (first 10*10 inputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", h_matB[loop1][loop2]);
    }

    double* d_matA;   hipMalloc(&d_matA, size);
    double* d_matB;   hipMalloc(&d_matB, size);
    double* d_matC;   hipMalloc(&d_matC, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_matA, h_matA, size,hipMemcpyHostToDevice);
    hipMemcpy(d_matB, h_matB, size,hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 threadsPerBlock = (16,16);
    dim3 blocksPerGrid ((N + threadsPerBlock.x - 1) /threadsPerBlock.x,(N + threadsPerBlock.y - 1) /threadsPerBlock.y);
    MatrixMulKernel_row_maj<<<blocksPerGrid, threadsPerBlock>>>(d_matA,d_matB, d_matC, N);

     // h_C contains the result in host memory
    hipMemcpy(h_matC, d_matC, size,hipMemcpyDeviceToHost);

    printf("\nMatrix C (first 10*10 outputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", h_matC[loop1][loop2]);
    }
      

    // Log outputs
    printf("\nWritting to file assignment_2_1_out as Mat C");
    print_matrix_to_file(h_matC,N,N);

    // Free device memory
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);

    // Free host memory
    free(h_matA);
    free(h_matB);
    free(h_matC);
    return 0;
}