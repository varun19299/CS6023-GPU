
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define TILE_WIDTH (16)

void fill_matrix(double *mat, unsigned numRows, unsigned numCols)
{
    for(unsigned i=0; i < numRows; i++)
       for(unsigned j=0; j < numCols; j++)
       {
         mat[i*numCols + j] = i*2.1f + j*3.2f;
       }
}

void print_matrix_to_file(double *mat, unsigned numRows, unsigned numCols)
{
  const char *fname = "assignment2_4_out";
  FILE *f = fopen(fname, "w");
  for(unsigned i=0; i < numRows; i++)
    {
        for(unsigned j=0; j < numCols; j++)
        fprintf(f,"%4.4f ", mat[i*numCols + j]);
        fprintf(f,"\n");
    }
    fclose(f); }

//template<int TILE_WIDTH>
__global__ void MatrixMulKernel_col_maj(double* M, double* N, double* Q, int Width) { 
    //extern __shared__ double buffer[];
    //double *ds_M = &buffer[0];
    //double *ds_N = &buffer[Width*Width];

    __shared__ double ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ double ds_N[TILE_WIDTH][TILE_WIDTH];

    // Generate IDs
    double Pvalue=0;
    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int Row = by * blockDim.y + ty;
    int Col = bx * blockDim.x + tx;

    
    // Loop over the M and N tiles required to compute the P element
    for (int p = 0; p < (Width)/TILE_WIDTH; ++p) {
        if ( (Row < Width) && (tx + p*TILE_WIDTH) < Width){
        // Collaborative loading of M and N tiles into shared memory
        ds_M[ty][tx] = M[Row*Width + p*TILE_WIDTH+tx];
        }
        else{
            ds_M[ty][tx]=0.0;
        }
        if ( (Col < Width) && (ty + p*TILE_WIDTH) < Width){
        ds_N[ty][tx] = N[(p*TILE_WIDTH+ty)*Width + Col];
        }
        else{
            ds_N[ty][tx]=0.0;
        }
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i){
            Pvalue += ds_M[ty][i] * ds_N[i][tx];
            
        }
        __syncthreads();
        
    }
    
    if ((Row < Width) && (Col < Width)){
    Q[Row*Width+Col] = Pvalue;
    }
}

int main(int argc,char **argv) {
    int N;
    N=8192;
    int loop1, loop2; // loop variables
    float time_spent;

    size_t size = N *N* sizeof(double);

    double*h_matA = (double*)malloc(size);
    double*h_matB = (double*)malloc(size);
    double*h_matC = (double*)malloc(size); // result

    fill_matrix(h_matA,N,N);
    fill_matrix(h_matB,N,N);

    printf("\nMatrix A (first 10*10 inputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matA + N*loop1 + loop2));
        printf("\n");
    }

    printf("\n\nMatrix B (first 10*10 inputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matB + N*loop1 + loop2));
        printf("\n");
    }

    double* d_matA;   hipMalloc(&d_matA, size);
    double* d_matB;   hipMalloc(&d_matB, size);
    double* d_matC;   hipMalloc(&d_matC, size);

    //GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_matA, h_matA, size,hipMemcpyHostToDevice);
    hipMemcpy(d_matB, h_matB, size,hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 threadsPerBlock(TILE_WIDTH,TILE_WIDTH);
    dim3 blocksPerGrid ((N + threadsPerBlock.x-1) /threadsPerBlock.x,(N + threadsPerBlock.y-1) /threadsPerBlock.y);

    hipEventRecord(start, 0);
    MatrixMulKernel_col_maj<<<blocksPerGrid, threadsPerBlock>>>(d_matA,d_matB, d_matC, N);
    //cudaDeviceSynchronize();//To synchronize the device
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_spent, start, stop);
    printf("\nTime spent in col maj %f\n",time_spent);

    // h_C contains the result in host memory
    hipMemcpy(h_matC, d_matC, size,hipMemcpyDeviceToHost);

    printf("\n\nMatrix C (first 10*10 outputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matC + N*loop1 + loop2));
        printf("\n");
    }

    // Log outputs
    printf("\nWritting to file assignment_2_1_out as Mat C");
    print_matrix_to_file(h_matC,N,N);

    // Free device memory
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);

    // Free host memory
    free(h_matA);
    free(h_matB);
    free(h_matC);
    return 0;
}