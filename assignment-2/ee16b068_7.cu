
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

void fill_matrix(double *mat, unsigned numRows, unsigned numCols)
{
    for(unsigned i=0; i < numRows; i++)
       for(unsigned j=0; j < numCols; j++)
       {
         mat[i*numCols + j] = i*2.1f + j*3.2f;
       }
}

void print_matrix_to_file(double *mat, unsigned numRows, unsigned numCols)
{
  const char *fname = "assignment2_7_out";
  FILE *f = fopen(fname, "w");
  for(unsigned i=0; i < numRows; i++)
    {
        for(unsigned j=0; j < numCols; j++)
        fprintf(f,"%4.4f ", mat[i*numCols + j]);
        fprintf(f,"\n");
    }
    fclose(f); }

__global__ void MatrixMulKernel_col_maj(double* M, double* N, double* P, int M_r, int N_c, int M_c, int TILE_WIDTH) { 
    extern __shared__ double buffer[];
    double *ds_M = &buffer[0];
    double *ds_N = &buffer[TILE_WIDTH*TILE_WIDTH];

    //__shared__ double ds_M[TILE_WIDTH][TILE_WIDTH];
    //__shared__ double ds_N[TILE_WIDTH][TILE_WIDTH];

    // Generate IDs
    double Pvalue=0;
    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int Row = by * blockDim.y + ty;
    int Col = bx * blockDim.x + tx;

    
    // Loop over the M and N tiles required to compute the P element
    for (int p = 0; p < (M_c)/TILE_WIDTH; ++p) {
        if ( (Row < M_r) && (tx + p*TILE_WIDTH) < M_c){
        // Collaborative loading of M and N tiles into shared memory
        ds_M[ty*TILE_WIDTH + tx] = M[Row*M_c + p*TILE_WIDTH+tx];
        }
        else{
            ds_M[ty*TILE_WIDTH + tx]=0.0;
        }
        if ( (Col < N_c) && (ty + p*TILE_WIDTH) < M_c){
            ds_N[ty*TILE_WIDTH + tx] = N[(p*TILE_WIDTH+ty)*N_c + Col];
        }
        else{
            ds_N[ty*TILE_WIDTH + tx]=0.0;
        }
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i){
            Pvalue += ds_M[ty*TILE_WIDTH + i] * ds_N[i*TILE_WIDTH + tx];
            
        }
        __syncthreads();
        
    }
    
    if ((Row < M_r) && (Col < N_c)){
    P[Row*N_c+Col] = Pvalue;
    }
}

int main(int argc,char **argv) {
    int M_r,M_c,N_c;
    int TILE_WIDTH_ll[4], TILE_WIDTH;
    float time_spent_ll[4], time_spent;
    
    int loop,loop1, loop2,min; // loop variables

    M_r=8192;
    M_c=16384;
    N_c=32768;

    size_t size1 = M_r *M_c* sizeof(double);
    size_t size2 = M_c *N_c* sizeof(double);
    size_t size3 = M_r *N_c* sizeof(double);

    double*h_matA = (double*)malloc(size1);
    double*h_matB = (double*)malloc(size2);
    double*h_matC = (double*)malloc(size3); // result

    fill_matrix(h_matA,M_r,M_c);
    fill_matrix(h_matB,M_c,N_c);

    for (loop = 0; loop<4; loop++){
        TILE_WIDTH_ll[loop]=pow(2,2+loop);
    }

    printf("\nMatrix A (first 10*10 inputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matA + M_c*loop1 + loop2));
        printf("\n");
    }

    printf("\n\nMatrix B (first 10*10 inputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matB + N_c*loop1 + loop2));
        printf("\n");
    }

    double* d_matA;   hipMalloc(&d_matA, size1);
    double* d_matB;   hipMalloc(&d_matB, size2);
    double* d_matC;   hipMalloc(&d_matC, size3);

    //GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_matA, h_matA, size1,hipMemcpyHostToDevice);
    hipMemcpy(d_matB, h_matB, size2,hipMemcpyHostToDevice);

    for (loop =0;loop < 4; loop++){

        TILE_WIDTH=TILE_WIDTH_ll[loop];
        
        // Invoke kernel
        dim3 threadsPerBlock (TILE_WIDTH,TILE_WIDTH,1);
        dim3 blocksPerGrid ((M_r + threadsPerBlock.x) /threadsPerBlock.x,(N_c + threadsPerBlock.y) /threadsPerBlock.y,1);

        size_t blocksize = 2 * TILE_WIDTH * TILE_WIDTH;

        hipEventRecord(start, 0);
        MatrixMulKernel_col_maj<<<blocksPerGrid, threadsPerBlock, sizeof(double)*blocksize>>>(d_matA,d_matB, d_matC, M_r,N_c,M_c, TILE_WIDTH);
        //cudaDeviceSynchronize();//To synchronize the device
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_spent, start, stop);
        printf("\nTime spent in col maj for tile %d x %d %f\n",TILE_WIDTH,TILE_WIDTH,time_spent);

        time_spent_ll[loop]=time_spent;
        
        // h_C contains the result in host memory
        hipMemcpy(h_matC, d_matC, size3,hipMemcpyDeviceToHost);

        printf("\n\nMatrix C (first 10*10 outputs)\n");
        for(loop1 = 0; loop1 < 10; loop1++){
            for (loop2=0;loop2 < 10; loop2++)
                printf("%f ", *(h_matC + N_c*loop1 + loop2));
            printf("\n");
        }
    }
    
    min=0;
    for ( loop = 1 ; loop < 4 ; loop++ ) 
    {
        if ( time_spent_ll[loop] < time_spent_ll[min] ) 
        {
           min = loop;
        }
    } 

    float min_time;
    int tile;

    min_time=time_spent_ll[min];
    tile=TILE_WIDTH_ll[min];


    printf("For the configuration of %d x %d multiplied by %d x %d \n",M_r,M_c,M_c,N_c);
    printf("Optimal time is %f, threads per block is %d x %d, tile size is %d x %d blocks per grid is %d x %d.",
    min_time,tile,tile,tile,tile,(M_r + tile) / tile,(N_c + tile) /tile);

    // Log outputs
    printf("\nWritting to file assignment_2_1_out as Mat C");
    print_matrix_to_file(h_matC,M_r,N_c);

    // Free device memory
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);

    // Free host memory
    free(h_matA);
    free(h_matB);
    free(h_matC);
    return 0;
}