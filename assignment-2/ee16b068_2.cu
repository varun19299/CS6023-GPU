
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

void fill_matrix(double *mat, unsigned numRows, unsigned numCols)
{
    for(unsigned i=0; i < numRows; i++)
       for(unsigned j=0; j < numCols; j++)
       {
         mat[i*numCols + j] = i*2.1f + j*3.2f;
       }
}

void print_matrix_to_file(double *mat, unsigned numRows, unsigned numCols)
{
  const char *fname = "assignment2_1_out";
  FILE *f = fopen(fname, "w");
  for(unsigned i=0; i < numRows; i++)
  {
     for(unsigned j=0; j < numCols; j++)
     fprintf(f,"%4.4f ", mat[i*numCols + j]);
     fprintf(f,"\n");
}
fclose(f); }

__global__ void MatrixMulKernel_col_maj(double* M, double* N, double* P, int Width) { 
    // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x+threadIdx.x; 
    
    if ((Row < Width) && (Col < Width)) {
            float Pvalue = 0;
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[Row*Width+k]*N[k*Width+Col];
        }
            P[Row*Width+Col] = Pvalue;
        }
    }

__global__ void MatrixMulKernel_row_maj(double* M, double* N, double* P, int Width) { 
    // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y+threadIdx.x;
    // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x+threadIdx.y; 
    
    if ((Row < Width) && (Col < Width)) {
            float Pvalue = 0;
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[Row*Width+k]*N[k*Width+Col];
        }
            P[Row*Width+Col] = Pvalue;
        }
    }

int main(int argc,char **argv) {
    int N = 8192;
    size_t size = N *N* sizeof(double);

    int thread_dim_ll[8];
    int thread_dim;

    double*h_matA = (double*)malloc(size);
    double*h_matB = (double*)malloc(size);
    double*h_matC = (double*)malloc(size); // result

    int loop, loop1, loop2; // loop variables
    float time_spent;

    fill_matrix(h_matA,N,N);
    fill_matrix(h_matB,N,N);

    printf("Thread dims\n");
    for (loop=0;loop<8;loop++){
        thread_dim_ll[loop]=pow(2,2+loop);
    }

    printf("\nMatrix A (first 10*10 inputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matA + N*loop1 + loop2));
    }

    printf("\n\nMatrix B (first 10*10 inputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matB + N*loop1 + loop2));
    }

    double* d_matA;   hipMalloc(&d_matA, size);
    double* d_matB;   hipMalloc(&d_matB, size);
    double* d_matC;   hipMalloc(&d_matC, size);

    //GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_matA, h_matA, size,hipMemcpyHostToDevice);
    hipMemcpy(d_matB, h_matB, size,hipMemcpyHostToDevice);

    for (loop = 0; loop < 8; loop++){

        //thread dim
        thread_dim=thread_dim_ll[loop];
        // Invoke kernel
        dim3 threadsPerBlock = (thread_dim,thread_dim);
        dim3 blocksPerGrid ((N + threadsPerBlock.x - 1) /threadsPerBlock.x,(N + threadsPerBlock.y - 1) /threadsPerBlock.y);

        hipEventRecord(start, 0);
        MatrixMulKernel_col_maj<<<blocksPerGrid, threadsPerBlock>>>(d_matA,d_matB, d_matC, N);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_spent, start, stop);
        printf("\nTime spent in col maj %f with threadsPerBlock %d \n",time_spent,thread_dim);

    }
    // h_C contains the result in host memory
    hipMemcpy(h_matC, d_matC, size,hipMemcpyDeviceToHost);

    printf("\n\nMatrix C (first 10*10 outputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matC + N*loop1 + loop2));
    }
      

    // Log outputs
    printf("\nWritting to file assignment_2_1_out as Mat C");
    print_matrix_to_file(h_matC,N,N);

    // Free device memory
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);

    // Free host memory
    free(h_matA);
    free(h_matB);
    free(h_matC);
    return 0;
}