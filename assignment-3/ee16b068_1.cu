#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#define MAXWORDS 20000

bool checkWord(word){
    // Check if word meets, else pre-process
}

__global__ void MatrixMulKernel_col_maj(double* M, double* N, double* P, int Width) { 
    // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x+threadIdx.x; 
    
    if ((Row < Width) && (Col < Width)) {
            float Pvalue = 0;
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[Row*Width+k]*N[k*Width+Col];
        }
            P[Row*Width+Col] = Pvalue;
        }
    }

__global__ void MatrixMulKernel_row_maj(double* M, double* N, double* P, int Width) { 
    // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y+threadIdx.x;
    // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x+threadIdx.y; 
    
    if ((Row < Width) && (Col < Width)) {
            float Pvalue = 0;
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[Row*Width+k]*N[k*Width+Col];
        }
            P[Row*Width+Col] = Pvalue;
        }
    }

int main(int argc,char **argv) {
    int N = 8192;
    size_t size = N *N* sizeof(double);

    double*h_matA = (double*)malloc(size);
    double*h_matB = (double*)malloc(size);
    double*h_matC = (double*)malloc(size); // result

    int loop1; int loop2; // loop variables
    float time_spent;

    fill_matrix(h_matA,N,N);
    fill_matrix(h_matB,N,N);

    printf("\nMatrix A (first 10*10 inputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matA + N*loop1 + loop2));
    }

    printf("\n\nMatrix B (first 10*10 inputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matB + N*loop1 + loop2));
    }

    double* d_matA;   hipMalloc(&d_matA, size);
    double* d_matB;   hipMalloc(&d_matB, size);
    double* d_matC;   hipMalloc(&d_matC, size);

    //GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_matA, h_matA, size,hipMemcpyHostToDevice);
    hipMemcpy(d_matB, h_matB, size,hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 threadsPerBlock = (16,16);
    dim3 blocksPerGrid ((N + threadsPerBlock.x - 1) /threadsPerBlock.x,(N + threadsPerBlock.y - 1) /threadsPerBlock.y);

    hipEventRecord(start, 0);
    MatrixMulKernel_col_maj<<<blocksPerGrid, threadsPerBlock>>>(d_matA,d_matB, d_matC, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_spent, start, stop);
    printf("\nTime spent in col maj %f\n",time_spent);

    // h_C contains the result in host memory
    hipMemcpy(h_matC, d_matC, size,hipMemcpyDeviceToHost);

    printf("\n\nMatrix C via col major (first 10*10 outputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matC + N*loop1 + loop2));
    }

    hipEventRecord(start, 0);
    MatrixMulKernel_row_maj<<<blocksPerGrid, threadsPerBlock>>>(d_matA,d_matB, d_matC, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_spent, start, stop);
    printf("\nTime spent in row maj %f\n",time_spent);

    // h_C contains the result in host memory
    hipMemcpy(h_matC, d_matC, size,hipMemcpyDeviceToHost);

    printf("\n\nMatrix C via row major (first 10*10 outputs)\n");
    for(loop1 = 0; loop1 < 10; loop1++){
        for (loop2=0;loop2 < 10; loop2++)
            printf("%f ", *(h_matC + N*loop1 + loop2));
    }
      

    // Log outputs
    printf("\nWritting to file assignment_2_1_out as Mat C");
    print_matrix_to_file(h_matC,N,N);

    // Free device memory
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);

    // Free host memory
    free(h_matA);
    free(h_matB);
    free(h_matC);
    return 0;
}
